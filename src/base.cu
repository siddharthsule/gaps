#include "base.cuh"

// Sync Device and Check for Errors
void syncGPUAndCheck(const char *operation) {
  // synchronize with the device
  hipDeviceSynchronize();

  // check for an error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message
    std::cerr << "CUDA error @" << operation << ": "
              << hipGetErrorString(error) << std::endl;

    // abort the program
    std::exit(EXIT_FAILURE);
  }
}

// Debug messages
__host__ __device__ void DEBUG_MSG(const char *message) {
  if (debug) {
    printf("DEBUG: %s\n", message);
  }
}