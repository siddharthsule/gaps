#include "base.cuh"

void syncGPUAndCheck(const char *operation) {
  // synchronize with the device
  hipDeviceSynchronize();

  // check for an error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message
    std::cerr << "CUDA error @" << operation << ": "
              << hipGetErrorString(error) << std::endl;

    // abort the program
    std::exit(EXIT_FAILURE);
  }
}