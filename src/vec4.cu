#include "hip/hip_runtime.h"
#include "vec4.cuh"

// Get Method to Obtain Attribute Value
double Vec4::operator[](int i) const {
    switch (i) {
        case 0:
            return E;
        case 1:
            return px;
        case 2:
            return py;
        case 3:
            return pz;
        default:
            // CUDA does not support exceptions, so we just return 0
            return 0;
    }
}

// Print a Column Vector with the attributes
std::ostream& operator<<(std::ostream& os, const Vec4& v) {
    os << "(" << v.E << "," << v.px << "," << v.py << "," << v.pz << ")";
    return os;
}

// Simple Mathematics with Four vectors
Vec4 Vec4::operator+(const Vec4& v) const {
    return Vec4(E+v.E, px+v.px, py+v.py, pz+v.pz);
}

Vec4 Vec4::operator-() const {
    return Vec4(-E, -px, -py, -pz);
}

Vec4 Vec4::operator-(const Vec4& v) const {
    return Vec4(E-v.E, px-v.px, py-v.py, pz-v.pz);
}

// Multiplication (and Dot Product)
double Vec4::operator*(const Vec4& v) const {
    return E*v.E-px*v.px-py*v.py-pz*v.pz;
}

Vec4 Vec4::operator*(double v) const {
    return Vec4(E*v, px*v, py*v, pz*v);
}

// Division
Vec4 Vec4::operator/(double v) const {
    return Vec4(E/v, px/v, py/v, pz/v);
}

// Magnitude of the Vector
double Vec4::M2() const {
    return (*this)*(*this);
}

double Vec4::M() const {
    double m2 = M2();
    return m2 > 0 ? sqrt(m2) : 0;
}

double Vec4::P2() const {
    return px*px + py*py + pz*pz;
}

double Vec4::P() const {
    double p2 = P2();
    return p2 > 0 ? sqrt(p2) : 0;
}

double Vec4::PT2() const {
    return px*px + py*py;
}

double Vec4::PT() const {
    double pt2 = PT2();
    return pt2 > 0 ? sqrt(pt2) : 0;
}

double Vec4::Theta() const {
    double p = P();
    return p != 0 ? acos(pz/p) : 0;
}

double Vec4::Phi() const {
    if (px == 0 && py == 0) {
        return 0.0;
    } else {
        return atan2(py, px);
    }
}

double Vec4::Rapidity() const {
    double denominator = (E - pz);
    return denominator != 0 ? 0.5 * log((E + pz)/denominator) : 0;
}

double Vec4::Eta() const {
    double theta = Theta();
    return - log(tan(theta/2.));
}

Vec4 Vec4::Cross(const Vec4& v) const {
    return Vec4(0.0,
                py*v.pz - pz*v.py,
                pz*v.px - px*v.pz,
                px*v.py - py*v.px);
}

Vec4 Vec4::Boost(const Vec4& v) const {
    double rsq = M();
    double v0 = (E*v.E - px*v.px - py*v.py - pz*v.pz)/rsq;
    double c1 = (v.E + v0)/(rsq + E);
    return Vec4(v0,
                v.px - c1*px,
                v.py - c1*py,
                v.pz - c1*pz);
}

Vec4 Vec4::BoostBack(const Vec4& v) const {
    double rsq = M();
    double v0 = (E*v.E + px*v.px + py*v.py + pz*v.pz)/rsq;
    double c1 = (v.E + v0)/(rsq + E);
    return Vec4(v0,
                v.px + c1*px,
                v.py + c1*py,
                v.pz + c1*pz);
}
