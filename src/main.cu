#include <fstream>
#include <iostream>
#include <tuple>
#include <vector>

#include "matrix.cuh"
#include "particle.cuh"
#include "vec4.cuh"

// Jet and Event Shape Analysis
#include "durham.h"
#include "eshape.h"

// Dipole Shower
#include "qcd.h"
#include "shower.h"

Event *gpuLOME(const int N) {
  // Allocate memory for a Matrix object on the device
  Matrix *d_matrix;
  hipMalloc(&d_matrix, sizeof(Matrix));

  // Set up the device Matrix object
  matrixSetupKernel<<<1, 1>>>(d_matrix, 0.118, 91.2);

  // Host and Device Variables
  Event *h_pd, *d_pd;

  h_pd = (Event *)malloc(N * sizeof(Event));
  hipMalloc(&d_pd, N * sizeof(Event));

  // Generate the LO Matrix Elements
  loPointKernel<<<(N + 255) / 256, 256>>>(d_matrix, d_pd, N);

  // Copy the results back to the host
  hipMemcpy(h_pd, d_pd, N * sizeof(Event), hipMemcpyDeviceToHost);

  // Free Memory
  hipFree(d_pd);

  return h_pd;
}

// Validation of Result Data
// Temporrily here, will be moved to a test suite
// Not in particle.cuh, which is now just a header file
bool IsEventCheckValid(const Event& ev) {
  Vec4 psum = Vec4();

  std::vector<int> csum(100, 0);

  for (int i = 0; i < ev.GetSize(); i++) {
    Particle p = ev.GetParton(i);

    Vec4 pmom = p.GetMom();
    int pcol = p.GetCol();
    int pAntiCol = p.GetAntiCol();

    psum = psum + pmom;

    if (pcol > 0) {
      csum[pcol] += 1;
    }

    if (pAntiCol > 0) {
      csum[pAntiCol] -= 1;
    }
  }

  bool pcheck = (psum[0] < 1e-12 && psum[1] < 1e-12 && psum[2] < 1e-12 &&
                 psum[3] < 1e-12);
  if (!pcheck) {
    std::cout << psum << std::endl;
  }

  bool ccheck = true;
  for (int i = 0; i < 100; i++) {
    if (csum[i] != 0) {
      std::cout << "Colour " << i << " is not conserved." << std::endl;
      ccheck = false;
      break;
    }
  }

  return pcheck && ccheck;
}

int runGenerator(const int &N, const std::string &filename = "test.yoda") {
  DAnalysis da;
  EAnalysis ea;

  AlphaS as(91.1876, 0.1181);
  Shower sh(1., as);

  Event *events = gpuLOME(N);

  for (int i = 0; i < N; i++) {

    Event ev = events[i];

    double t = (ev.GetParton(0).GetMom() + ev.GetParton(1).GetMom()).M2();
    sh.Run(ev, t);

    //std::cout << "Event " << i << " has " << ev.GetSize() << " partons" <<
    //std::endl; for (auto& p : ev.partons) { std::cout << p.GetPid() << " ";}
    

    if (IsEventCheckValid(ev)) {
      if (ev.GetSize() > 4) {
        da.Analyze(ev);
        ea.Analyze(ev);
      }
    } else {
      std::cout << "Event failed validation" << std::endl;
    }

    if (i % 1000 == 0) {
      std::cout << "Running Event " << i << std::endl;
    }
  }

  da.Finalize(filename);
  ea.Finalize(filename);

  return 0;
}

int main(int argc, char *argv[]) {
  int N = argc > 1 ? atoi(argv[1]) : 10000;
  std::string filename = "output.yoda";
  int remover = std::remove("outpt.yoda");
  runGenerator(N, filename);
  return 0;
}
