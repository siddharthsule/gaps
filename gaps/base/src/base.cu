#include "base.cuh"

// sync device and check for errors
void sync_gpu_and_check(const char *operation) {
  // synchronize with the device
  hipDeviceSynchronize();

  // check for an error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // print the cuda error message
    std::cerr << "CUDA error @" << operation << ": "
              << hipGetErrorString(error) << std::endl;

    // abort the program
    std::exit(EXIT_FAILURE);
  }
}

// debug messages
__host__ __device__ void debug_msg(const char *message) {
  if (debug) {
    printf("debug: %s\n", message);
  }
}