#include "hip/hip_runtime.h"
#include "matrix.cuh"

// -----------------------------------------------------------------------------
// constructor

__device__ void matrix::setup(bool nlo, double root_s) {
  this->nlo = nlo;
  this->root_s = root_s;
  this->s = pow(root_s, 2.);
  this->s_hat = s;
  this->mz2 = pow(mz, 2.);
  this->gz2 = pow(gz, 2.);
  this->alpha = 1. / 128.802;
  this->sin2tw = 0.22293;
  this->amin = 1.e-10;
  this->ye = 0.5;
  this->ze = 0.01;
  this->ws = 0.25;
}

// kernel to set up the matrix object on the device
__global__ void matrix_setup_kernel(matrix *matrix, bool nlo, double root_s) {
  matrix->setup(nlo, root_s);
}

// -----------------------------------------------------------------------------
// main

// function to generate the lo matrix elements + momenta
void calc_lome(thrust::device_vector<event> &d_events, bool nlo, double root_s,
               double asmz, int blocks, int threads) {
  /**
   * @brief wrap
   */

  // number of events - can get from d_events.size()
  int n = d_events.size();

  // allocate memory for a matrix object on the device
  matrix *d_matrix;
  hipMalloc(&d_matrix, sizeof(matrix));

  // set up the device matrix object
  debug_msg("running @matrix_setup_kernel");
  matrix_setup_kernel<<<1, 1>>>(d_matrix, nlo, root_s);
  sync_gpu_and_check("matrix_setup_kernel");

  // set up the device alpha_s calculator
  alpha_s *d_as;
  hipMalloc(&d_as, sizeof(alpha_s));
  as_setup_kernel<<<1, 1>>>(d_as, mz, asmz);
  sync_gpu_and_check("as_setup_kernel");

  // LEP LO
  if (!nlo) {
    debug_msg("running @lep_lo");
    lep_lo<<<blocks, threads>>>(d_matrix,
                                thrust::raw_pointer_cast(d_events.data()), n);
    sync_gpu_and_check("lep_lo");
  }

  // LEP NLO
  else {
    debug_msg("running @lep_lo and @lep_nlo");
    lep_lo<<<blocks, threads>>>(d_matrix,
                                thrust::raw_pointer_cast(d_events.data()), n);
    sync_gpu_and_check("lep_lo");
    lep_nlo<<<blocks, threads>>>(d_matrix, d_as,
                                 thrust::raw_pointer_cast(d_events.data()), n);
    sync_gpu_and_check("lep_nlo");
  }

  return;
}