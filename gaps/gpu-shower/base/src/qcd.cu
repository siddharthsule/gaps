#include "hip/hip_runtime.h"
#include "qcd.cuh"

// constructor
__device__ alpha_s::alpha_s(double mz, double asmz, int order, double mb,
                            double mc)
    : order(order),
      mc2(mc * mc),
      mb2(mb * mb),
      mz2(mz * mz),
      asmz(asmz),
      asmb((*this)(mb2)),
      asmc((*this)(mc2)) {}

// setup
__device__ void alpha_s::setup(double mz, double asmz, int order, double mb,
                               double mc) {
  this->order = order;
  this->mc2 = mc * mc;
  this->mb2 = mb * mb;
  this->mz2 = mz * mz;
  this->asmz = asmz;
  this->asmb = (*this)(mb2);
  this->asmc = (*this)(mc2);
}

// beta and alpha s functions
__device__ double alpha_s::beta0(int nf) const {
  /**
   * @brief calculate the beta function at order 0
   *
   * @param nf the number of flavours
   * @return the beta function
   */

  return (11. / 6. * k_ca) - (2. / 3. * k_tr * nf);
}

__device__ double alpha_s::beta1(int nf) const {
  /**
   * @brief calculate the beta function at order 1
   *
   * @param nf the number of flavours
   * @return the beta function
   */

  return (17. / 6. * k_ca * k_ca) - ((5. / 3. * k_ca + k_cf) * k_tr * nf);
}

__device__ double alpha_s::as0(double t) const {
  /**
   * @brief calculate the strong coupling constant at order 0
   *
   * @param t the scale
   * @return the strong coupling constant
   */

  double tref, asref, b0;

  // Threshold Matching OFF
  // if (t >= mb2) {
  tref = mz2;
  asref = asmz;
  b0 = beta0(5) / (2. * M_PI);
  // } else if (t >= mc2) {
  //   tref = mb2;
  //   asref = asmb;
  //   b0 = beta0(4) / (2. * M_PI);
  // } else {
  //   tref = mc2;
  //   asref = asmc;
  //   b0 = beta0(3) / (2. * M_PI);
  // }

  return 1. / (1. / asref + b0 * log(t / tref));
}

__device__ double alpha_s::as1(double t) const {
  /**
   * @brief calculate the strong coupling constant at order 1
   *
   * @param t the scale
   * @return the strong coupling constant
   */

  double tref, asref, b0, b1, w;

  // Threshold Matching OFF
  // if (t >= mb2) {
  tref = mz2;
  asref = asmz;
  b0 = beta0(5) / (2. * M_PI);
  b1 = beta1(5) / pow(2. * M_PI, 2);
  // } else if (t >= mc2) {
  //   tref = mb2;
  //   asref = asmb;
  //   b0 = beta0(4) / (2. * M_PI);
  //   b1 = beta1(4) / pow(2. * M_PI, 2);
  // } else {
  //   tref = mc2;
  //   asref = asmc;
  //   b0 = beta0(3) / (2. * M_PI);
  //   b1 = beta1(3) / pow(2. * M_PI, 2);
  // }

  w = 1. + b0 * asref * log(t / tref);
  return asref / w * (1. - b1 / b0 * asref * log(w) / w);
}

__device__ double alpha_s::operator()(double t) {
  /**
   * @brief wrapper/call operator for the strong coupling constant. This
   * function will calculate the strong coupling constant at the given scale,
   * the order is determined by the member variable order
   *
   * @param t the scale
   * @return the strong coupling constant
   */

  if (order == 0) {
    return as0(t);
  } else {
    return as1(t);
  }
}

// set up kernel on the device
__global__ void as_setup_kernel(alpha_s *as, double mz, double asmz, int order,
                                double mb, double mc) {
  /**
   * @brief set up the alpha_s class on the device
   *
   * @param as the alpha_s class
   * @param mz the Z boson mass
   * @param asmz the strong coupling constant at the Z boson mass
   * @param order the order of the strong coupling constant
   * @param mb the bottom quark mass
   * @param mc the charm quark mass
   */

  as->setup(mz, asmz, order, mb, mc);
}

// calculate alpha_s on the device for one input
__global__ void as_value(alpha_s *as, double *as_val, double t) {
  /**
   * @brief calculate the strong coupling constant at a given scale
   *
   * @param as the alpha_s class
   * @param as_val the strong coupling constant
   * @param t the scale
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= 1) return;
  // ---------------------------------------------

  as_val[idx] = (*as)(t);
}