#include "hip/hip_runtime.h"
#include "shower.cuh"

// -----------------------------------------------------------------------------
// constructor

__device__ void shower::setup(double root_s, double t_c, double as_max) {
  this->t_c = t_c;
  this->as_max = as_max;
  this->j_max = 1.;
}

// kernel to set up the matrix object on the device
__global__ void shower_setup_kernel(shower *sh, double root_s, double t_c,
                                    double as_max) {
  /**
   * @brief Set up the shower object on the device
   *
   * @param sh The shower object
   * @param as The alpha_s object
   * @param root_s The root s energy
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= 1) return;
  // ---------------------------------------------
  sh->setup(root_s, t_c, as_max);
}

// -----------------------------------------------------------------------------
// preparing the shower

__global__ void prep_shower(event *events, bool nlo_matching, int n) {
  /**
   * @brief Prepares the shower for the event
   *
   * @param events The events to prepare
   * @param n The number of events
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  // ---------------------------------------------
  // Shower Preamble
  event &ev = events[idx];
  // ---------------------------------------------

  // NLO Matching does the first emission and sets the shower scale
  // to the first emission pT. If NLO Matching is off, we find the
  // smallest pT in the event and set the shower scale to that.
  if (!nlo_matching) {
    // set the starting shower scale
    double t_start = 10000000.;
    for (int i = 0; i < ev.get_size(); i++) {
      for (int j = 0; j < ev.get_size(); j++) {
        if (i == j) {
          continue;
        }

        if (!ev.get_particle(i).is_parton() ||
            !ev.get_particle(j).is_parton()) {
          continue;
        }

        double t =
            (ev.get_particle(i).get_mom() + ev.get_particle(j).get_mom()).m2();

        if (t < t_start) {
          t_start = t;
        }
      }
    }

    ev.set_shower_t(t_start);
    ev.set_shower_c(1);
  }
}

// -----------------------------------------------------------------------------

__global__ void select_winner_split_func(shower *shower, event *events, int n,
                                         double *winner) {
  /**
   * @brief Select the winner splitting in the event
   *
   * This function generates the highest transverse momentum splitting for every
   * possible dipoles in the event. It then chooses the winner emission from the
   * generated splittings, by picking the one with the highest transverse
   * momentum. This winner emission is then used in the veto step
   *
   * when you profile the code, you will notice that this is the process that
   * takes up half of the shower time. this method below is a first attempt at
   * parallelizing the process.
   *
   * @param shower The shower object
   * @param events The events to run the shower on
   * @param n The number of events
   * @param winner The array to store the winner variables
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  // ---------------------------------------------
  // Shower Preamble
  if (events[idx].has_shower_ended()) return;
  event &ev = events[idx];
  // ---------------------------------------------

  // default values
  double win_tt = shower->t_c;  // lowest possible value is cutoff
  int win_sf = 0;               // 0 = no splitting
  int win_ij = 0;
  int win_k = 0;
  double win_sijk = 0.;
  double win_zm = 0.;
  double win_zp = 0.;

  for (int ij = 0; ij < ev.get_size(); ij++) {
    for (int k = 0; k < ev.get_size(); k++) {
      // sanity check to ensure ij != k
      if (ij == k) {
        continue;
      }

      // Skip non-partons
      if (!ev.get_particle(ij).is_parton() || !ev.get_particle(k).is_parton()) {
        continue;
      }

      // need to check if ij and k are colour connected
      if (!ev.get_particle(ij).is_color_connected(ev.get_particle(k))) {
        continue;
      }

      // identical to all splitting functions
      double sijk =
          (ev.get_particle(ij).get_mom() + ev.get_particle(k).get_mom()).m2();
      if (sijk < 4. * shower->t_c) {
        continue;
      }

      double zp = 0.5 * (1. + sqrt(1. - 4. * shower->t_c / sijk));
      double zm = 1. - zp;
      if (zm < 0. || zp > 1. || zm > zp) {
        continue;
      }

      // get the splitting functions for the current partons
      int splittings[6];
      shower->get_possible_splittings(ev.get_particle(ij).get_pid(),
                                      splittings);

      // codes instead of object oriented approach!
      for (int sf : splittings) {
        // When a null code is encountered, we have reached the end of the
        // possible splittings, we can break out of the loop
        if (sf == -1) {
          break;
        }

        // calculate the integrated overestimate
        double c = shower->j_max * shower->as_max / (2. * M_PI) *
                   shower->sf_integral(zm, zp, sf);

        // calculate the evolution variable
        double tt = ev.get_shower_t() * pow(ev.gen_random(), 1. / c);

        // check if tt is greater than the current winner
        if (tt > win_tt) {
          win_tt = tt;
          win_sf = sf;
          win_ij = ij;
          win_k = k;
          win_sijk = sijk;
          win_zm = zm;
          win_zp = zp;
        }
      }
    }
  }

  // set the new shower t
  ev.set_shower_t(win_tt);

  // Also generate z, y and phi
  double z = shower->sf_generate_z(win_zm, win_zp, ev.gen_random(), win_sf);
  double y = win_tt / win_sijk / z / (1. - z);
  double phi = 2. * M_PI * ev.gen_random();

  // Set the winner variables (sf, ij, k, sijk, z, y, phi)
  winner[7 * idx] = static_cast<double>(win_sf);
  winner[7 * idx + 1] = static_cast<double>(win_ij);
  winner[7 * idx + 2] = static_cast<double>(win_k);
  winner[7 * idx + 3] = win_sijk;
  winner[7 * idx + 4] = z;
  winner[7 * idx + 5] = y;
  winner[7 * idx + 6] = phi;
}

// -----------------------------------------------------------------------------

__global__ void check_cutoff(event *events, shower *shower, int *d_completed,
                             int n) {
  /**
   * @brief Check if the shower has ended
   *
   * @param events The events to run the shower on
   * @param d_completed The number of completed events
   * @param cutoff The cutoff scale
   * @param n The number of events
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  // ---------------------------------------------
  // Shower Preamble
  if (events[idx].has_shower_ended()) return;
  event &ev = events[idx];
  // ---------------------------------------------

  // limit to one emission
  // if (ev.get_emissions() == 1) {
  //   ev.shower_has_ended(true);
  //   atomicAdd(d_completed, 1);  // increment the number of completed events
  //   return;
  // }

  /**
   * end shower if t < cutoff
   *
   * ev.get_shower_t() <= cutoff is equally valid
   * i just prefer this way because this way is
   * how we usually write it in literature
   */
  if (!(ev.get_shower_t() > shower->t_c)) {
    ev.shower_has_ended(true);
    atomicAdd(d_completed, 1);  // increment the number of completed events

    return;
  }
}

// -----------------------------------------------------------------------------

/**
 * PDF Ratio Calculation
 * ---------------------
 *
 * This is done in two steps:
 * - PDFs are evaluated for ij and i, see pdf.cuh
 * - Ratio is calculated in the veto_alg kernel
 */

// -----------------------------------------------------------------------------

__global__ void veto_alg(shower *shower, alpha_s *as, event *events, int n,
                         bool *accept_emission, double *winner) {
  /**
   * @brief The veto algorithm for the shower
   *
   * @param shower The shower object
   * @param as The alpha_s object
   * @param events The events to run the shower on
   * @param n The number of events
   * @param xf_a The PDF of the parton after emission
   * @param xf_b The PDF of the parton before emissions
   * @param accept_emission The array to store the acceptance of the emission
   * @param winner The array to store the winner emission data
   * @param d_evaluations The number of evaluations
   * @param d_overestimate_error The number of overestimate errors
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  // ---------------------------------------------
  // Shower Preamble
  if (events[idx].has_shower_ended()) return;
  event &ev = events[idx];
  // ---------------------------------------------

  // set to false, only set to true if accpeted
  accept_emission[idx] = false;

  // Get the shower evolution variable
  double t = ev.get_shower_t();

  // Get the winner variables (sf, ij, j, sijk, z, y, phi)
  int sf = static_cast<int>(winner[7 * idx]);
  // int ij = static_cast<int>(winner[7 * idx + 1]);
  // int k = static_cast<int>(winner[7 * idx + 2]);
  // double sijk = winner[7 * idx + 3];
  double z = winner[7 * idx + 4];
  double y = winner[7 * idx + 5];
  // double phi = winner[7 * idx + 6];

  // Check Phase Space is Valid
  if (z < 0. || z > 1. || y < 0. || y > 1.) {
    return;
  }

  // veto algorithm
  double f = (*as)(t)*shower->sf_value(z, y, sf) * (1. - y);
  double g = shower->as_max * shower->sf_estimate(z, sf) * shower->j_max;

  // Check for Negative f
  if (f < 0.) {
    return;
  }

  if (ev.gen_random() < f / g) {
    accept_emission[idx] = true;
  }

  return;
}

// -----------------------------------------------------------------------------

// do splitting
__global__ void do_splitting(shower *shower, event *events, int n,
                             bool *accept_emission, double *winner) {
  /**
   * @brief Do the splitting for the shower
   *
   * @param shower The shower object
   * @param events The events to run the shower on
   * @param n The number of events
   * @param accept_emission The array to store the acceptance of the emission
   * @param winner The array to store the winner emission data
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  // ---------------------------------------------
  // Shower Preamble
  if (events[idx].has_shower_ended()) return;
  event &ev = events[idx];
  // ---------------------------------------------

  // Do not run if the shower has ended
  if (!accept_emission[idx]) {
    return;
  }

  // Get the shower evolution variable
  double t = ev.get_shower_t();

  // Get the winner variables (sf, ij, j, sijk, z, y, phi)
  int sf = static_cast<int>(winner[7 * idx]);
  int ij = static_cast<int>(winner[7 * idx + 1]);
  int k = static_cast<int>(winner[7 * idx + 2]);
  // double sijk = winner[7 * idx + 3];
  double z = winner[7 * idx + 4];
  double y = winner[7 * idx + 5];
  double phi = winner[7 * idx + 6];

  // get flavs from kernel number
  int flavs[3];
  shower->sf_to_flavs(sf, flavs);

  // pi, pj, pk, pijt, pkt and kt
  vec4 moms[6] = {vec4(), vec4(), vec4(), vec4(), vec4(), vec4()};
  shower->make_kinematics(moms, z, y, phi, ev.get_particle(ij).get_mom(),
                          ev.get_particle(k).get_mom(), sf);

  // calculate the colours
  int colij[2] = {ev.get_particle(ij).get_col(),
                  ev.get_particle(ij).get_acol()};
  int colk[2] = {ev.get_particle(k).get_col(), ev.get_particle(k).get_acol()};
  int coli[2] = {0, 0};
  int colj[2] = {0, 0};
  shower->make_colours(ev.get_shower_c(), sf, flavs, colij, colk, coli, colj,
                       ev.gen_random());

  // modify splitter
  ev.set_particle_pid(ij, flavs[1]);
  ev.set_particle_mom(ij, moms[0]);
  ev.set_particle_col(ij, coli[0]);
  ev.set_particle_acol(ij, coli[1]);

  // modify recoiled spectator
  ev.set_particle_mom(k, moms[2]);

  // add emitted particle
  particle em = particle(flavs[2], moms[1], colj[0], colj[1]);
  ev.set_particle(ev.get_size(), em);

  // increment emissions (important) !!!!!
  ev.increment_emissions();

  return;
}

// -----------------------------------------------------------------------------

__global__ void check_too_many_particles(event *events, int n_emissions_max,
                                         int *d_too_many_particles,
                                         int *d_completed, int n) {
  /**
   * @brief Check if the event has too many particles
   *
   * @param events The events to run the shower on
   * @param d_too_many_particles The number of events with too many particles
   * @param n The number of events
   */
  // ---------------------------------------------
  // Kernel Preamble
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) return;
  // ---------------------------------------------
  // Shower Preamble
  if (events[idx].has_shower_ended()) return;
  event &ev = events[idx];
  // ---------------------------------------------

  // limit to max particles
  if (ev.get_size() == min(max_particles, ev.get_hard() + n_emissions_max)) {
    ev.shower_has_ended(true);
    atomicAdd(d_completed, 1);  // increment the number of completed events
    atomicAdd(d_too_many_particles, 1);  // to print later
    return;
  }
}

// -----------------------------------------------------------------------------

void run_shower(thrust::device_vector<event> &dv_events, double root_s,
                bool nlo_matching, bool do_partition, double t_c, double asmz,
                int n_emissions_max, int blocks, int threads) {
  /**
   * @brief Run the shower on the events
   *
   * @param dv_events The events to run the shower on
   * @param root_s The root s energy
   * @param nlo_matching Whether to do NLO matching
   * @param do_partition Whether to partition the events
   */

  // number of events - can get from d_events.size()
  event *d_events = thrust::raw_pointer_cast(dv_events.data());
  int n = dv_events.size();

  // set up the device alpha_s calculator
  alpha_s *d_as;
  hipMalloc(&d_as, sizeof(alpha_s));
  as_setup_kernel<<<1, 1>>>(d_as, mz, asmz);
  sync_gpu_and_check("as_setup_kernel");

  // Calculate as_max = as(t_c)
  double *d_as_max;
  hipMalloc(&d_as_max, sizeof(double));
  as_value<<<1, 1>>>(d_as, d_as_max, t_c);
  sync_gpu_and_check("as_value");
  double as_max;
  hipMemcpy(&as_max, d_as_max, sizeof(double), hipMemcpyDeviceToHost);

  // set up the shower
  shower *d_shower;
  hipMalloc(&d_shower, sizeof(shower));
  shower_setup_kernel<<<1, 1>>>(d_shower, root_s, t_c, as_max);
  sync_gpu_and_check("shower_setup_kernel");

  /**
   * Shower Variables - useful to store as collective
   *
   * t, c and end_shower stored in event, becuase they
   * are unique to each event, and not throwaway values
   * like these.
   *
   * Winner variables: (sf, ij, k, sijk, z, y, phi)
   * Stored in ONE array, so we make it 7 x n
   * Stored all as doubles, so static_cast<int> for sf, ij, k
   */
  thrust::device_vector<double> dv_winner(7 * n, 0.0);
  double *d_winner = thrust::raw_pointer_cast(dv_winner.data());

  // veto outcome
  thrust::device_vector<bool> dv_accept_emission(n, false);
  bool *d_accept_emission = thrust::raw_pointer_cast(dv_accept_emission.data());

  // ---------------------------------------------------
  // Analysis Variables

  // allocate device memory for completed events counter
  int *d_completed;
  hipMalloc(&d_completed, sizeof(int));
  hipMemset(d_completed, 0, sizeof(int));

  // allocate device memory to counts events that surpass max particles
  int *d_too_many_particles;
  hipMalloc(&d_too_many_particles, sizeof(int));
  hipMemset(d_too_many_particles, 0, sizeof(int));

  // store the number of time and finished events per cycle
  std::vector<double> time_per_cycle;
  std::vector<int> completed_per_cycle;

  // ---------------------------------------------------------------------------
  // prepare the shower

  debug_msg("running @prep_shower");
  prep_shower<<<blocks, threads>>>(d_events, nlo_matching, n);
  sync_gpu_and_check("prep_shower");

  // ---------------------------------------------------------------------------
  // run the shower

  // ----------------------------------------------------
  // start the clock to analyse the time per cycle
  auto start = std::chrono::high_resolution_clock::now();

  // dummy variables to store the time and difference
  auto end = std::chrono::high_resolution_clock::now();
  double diff = 0.;
  // ----------------------------------------------------

  // number of completed events and cycles
  int completed = 0;
  int cycle = 0;

  while (completed < n) {
    // run all the kernels here...

    // -------------------------------------------------------------------------
    // select the winner kernel

    debug_msg("running @select_winner_split_func");
    select_winner_split_func<<<blocks, threads>>>(d_shower, d_events, n,
                                                  d_winner);
    sync_gpu_and_check("select_winner_split_func");

    // -------------------------------------------------------------------------
    // check cutoff

    debug_msg("running @check_cutoff");
    check_cutoff<<<blocks, threads>>>(d_events, d_shower, d_completed, n);
    sync_gpu_and_check("check_cutoff");

    // -------------------------------------------------------------------------
    // veto algorithm

    debug_msg("running @veto_alg");
    veto_alg<<<blocks, threads>>>(d_shower, d_as, d_events, n,
                                  d_accept_emission, d_winner);
    sync_gpu_and_check("veto_alg");

    // -------------------------------------------------------------------------
    // splitting algorithm

    debug_msg("running @do_splitting");
    do_splitting<<<blocks, threads>>>(d_shower, d_events, n, d_accept_emission,
                                      d_winner);
    sync_gpu_and_check("do_splitting");

    // -------------------------------------------------------------------------
    // check if there are too many particles

    debug_msg("running @check_too_many_particles");
    check_too_many_particles<<<blocks, threads>>>(
        d_events, n_emissions_max, d_too_many_particles, d_completed, n);
    sync_gpu_and_check("check_too_many_particles");

    // -------------------------------------------------------------------------
    // import the number of completed events

    hipMemcpy(&completed, d_completed, sizeof(int), hipMemcpyDeviceToHost);
    cycle++;

    // -------------------------------------------------------------------------
    // store the number of completed events and time per cycle

    // until paper is published, we will use this
    completed_per_cycle.push_back(completed);
    std::cerr << "\rCompleted Events: " << completed << "/" << n << std::flush;

    // end the clock
    end = std::chrono::high_resolution_clock::now();
    diff = std::chrono::duration<double>(end - start).count();
    time_per_cycle.push_back(diff);

    // -------------------------------------------------------------------------
  }
  std::cout << std::endl;

  // print the number of events that surpassed the max particles
  int too_many_particles;
  hipMemcpy(&too_many_particles, d_too_many_particles, sizeof(int),
             hipMemcpyDeviceToHost);
  if (too_many_particles > 0) {
    if (max_particles < n_emissions_max) {
      std::cerr << "Warning: " << too_many_particles
                << " events surpassed the maximum number of particles"
                << std::endl;
      std::cerr << "Consider increasing max_particles, default: "
                << max_particles << std::endl;
    }
  }

  // ---------------------------------------------------------------------------
  // write completed_per_cycle to file
  std::ofstream file("gpu-cycles.dat");
  for (int i = 0; i < cycle; i++) {
    file << time_per_cycle[i] << ", " << n - completed_per_cycle[i]
         << std::endl;
  }

  // ---------------------------------------------------------------------------

  // free the memory
  hipFree(d_shower);
  hipFree(d_as);
  hipFree(d_completed);
  hipFree(d_too_many_particles);
}