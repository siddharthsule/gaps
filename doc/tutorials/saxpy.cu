#include "hip/hip_runtime.h"
// From https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/
#include <stdio.h>

// Large array, 2^20
#define N 1048576

// Kernel function to add the elements of two arrays
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

int main(void) {
  // Host input vectors
  float *h_x, *h_y, *d_x, *d_y;
  h_x = (float *)malloc(N * sizeof(float));
  h_y = (float *)malloc(N * sizeof(float));

  // Device input vectors
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  // Initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    h_x[i] = 1.0f;
    h_y[i] = 2.0f;
  }

  // Copy data from host to device
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  // Copy data from device to host
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  // Check for errors (all values should be 4.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = max(maxError, abs(y[i] - 4.0f));
  printf("Max error: %f\n", maxError);

  // Cleanup
  hipFree(d_x);
  hipFree(d_y);
  free(h_x);
  free(h_y);
}
