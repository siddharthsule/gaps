int main(void) {
  // DOES NOT RUN

  // Declare Variables
  // h_ = on host, d_ = on device
  int *h_c, d_c;

  // Allocate memory on the device
  // hipMalloc( Location of the Memory, Size of the Memory )
  hipMalloc((void**)&d_c, sizeof(int));

  // If h_c initialised, copy info from h_c to d_c
  // hipMemcpy( destination, host, numBytes, Direction )
  hipMemcpy(d_c, h_c, sizeof(int) hipMemcpyHostToDevice);

  // Kernel Configuration Parameters
  dim3 grid_size(1);
  dim3 block_size(1);

  // Launch the Kernel
  kernel<<<grid_size, block_size>>>(...);

  // Copy data back to host
  // hipMemcpy( destination, device, numBytes, Direction )
  hipMemcpy(h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

  // Deallocate Memory
  hipFree(d_c);
  free(h_c);

  return 0;
}
